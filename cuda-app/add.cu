/**
 * @file interposer.c
 * @brief Demonstrates using of libcudainterpose.c
 *
 * @date Feb 4, 2011
 * @author Magda S., magg@gatech.edu
 */

//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
		a[idx] = a[idx] * a[idx];
}

// main routine that executes on the host
// >>>>>>>>> change main() -> cuda_main()
int cuda_main(void) {
	float *a_h, *a_d; // Pointer to host & device arrays
	const int N = 10; // Number of elements in arrays
	size_t size = N * sizeof(float);

	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	a_h = (float *) malloc(size); // Allocate array on host
	hipMalloc((void **) &a_d, size); // Allocate array on device
	// Initialize host array and copy it to CUDA device
	for (int i = 0; i < N; i++)
		a_h[i] = (float) i;
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	// Do calculation on device:
	int block_size = 4;
	int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
	square_array <<< n_blocks, block_size >>> (a_d, N);
	// Retrieve result from device and store it in host array
	hipMemcpy(a_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost);
	// Print results
	for (int i = 0; i < N; i++)
		printf("%d %f\n", i, a_h[i]);
	// Cleanup
	free(a_h);
	hipFree(a_d);

	return 0;
}

int main(){
	// I wonder, if this cannot be done by Python, since this is administration
	// and python integrates with c, so I think it might be wiser
	// to use python for that. but maybe later, when the thinks will clarify
	// create GPU assembly
	// 0. init if not initialized
	// (not here outthere) the device - create in buStore the representations
	// of the physical devices
	// 1. specify how many GPU you need
	// 2. create as many vgpu (in terms of structures) as required
	// 3. wire vgpu to gpus
	// 4. gpu assembly
	// 5. some process needs to clean up after - but we do not worry about that

	// now call the cuda main, so our GA enabler library can sort out
	// which cuda call goes where
	cuda_main();

	return 0;
}
